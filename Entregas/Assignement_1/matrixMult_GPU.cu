#include "hip/hip_runtime.h"
#include "common.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <math.h>
#include <hip/hip_fp16.h>
#include <chrono>

using namespace std;

void printMatrix(float *mat, const int nx, const int ny){
  for (int iy = 0; iy < ny; iy++) {
    for (int ix = 0; ix < nx; ix++)
      cout << mat[ix] << " ";
    cout << endl;
    mat += nx;
  }

  return;
}

void initialData(float *ip, const int size){
    int i;
    for(i = 0; i < size; i++)
        ip[i] = i * 2;
        //ip[i] = (float)(rand() & 0xFF) / 10.0f;
    return;
}

// grid 2D block 1D
__global__ void multMatrixOnGPU2d1d(float *MatA, float *MatB, float *MatC, int nx, int ny) {

    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = blockIdx.y;

    //unsigned int col_position = idx % nx;
    //unsigned int row_position = (int)floorf ( (float)(idx / ny ));
    //unsigned int initial_col_mult = idx - col_position;

    unsigned int idx;
    if (ix < nx && iy < ny){
        idx = iy * nx + ix;
        unsigned int col_position = idx % nx;
        printf("Index en h_R es %d con fil y col %d %d\n", idx, iy, col_position);
        printf("En h_A comienza a multiplicar desde col %d fil %d\n", idx - col_position );
    }

    //float sum = 0.0;

    // if (ix < nx && iy < ny)
    //   for (int i = 0; i < nx; i++)
    //     sum = sum + MatA[initial_col_mult + i] * MatB[i * nx + row_position];
    //   MatC[idx] = sum;
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev), "Error device prop");
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    SAFE_CALL(hipSetDevice(dev), "Error setting device");

    // set up data size of matrix
    int nx = 1 << 2;
    int ny = 1 << 2;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);
    printf("Matrix size: nx %d ny %d\n", nx, ny);

    // malloc host memory
    float *h_A, *h_B, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    // initialize data at host side

    initialData(h_A, nxy);
    initialData(h_B, nxy);

    memset(gpuRef, 0, nBytes);

    // malloc device global memory
    float *d_MatA, *d_MatB, *d_MatC;
    SAFE_CALL(hipMalloc((void **)&d_MatA, nBytes), "Error allocating d_MatA");
    SAFE_CALL(hipMalloc((void **)&d_MatB, nBytes), "Error allocating d_MatB");
    SAFE_CALL(hipMalloc((void **)&d_MatC, nBytes), "Error allocating d_MatC");

    // transfer data from host to device
    SAFE_CALL(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice), "Error copying d_MatA");
    SAFE_CALL(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice), "Error copying d_MatB");

    // invoke kernel at host side
    int dimx = 128;
    dim3 block(dimx, 1);
    dim3 grid((nx + block.x - 1) / block.x, ny);

    //printMatrix(h_A, nx, ny);
    //printMatrix(h_B, nx, ny);

    auto start_cpu =  chrono::high_resolution_clock::now();
    multMatrixOnGPU2d1d<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel");
    auto end_cpu =  chrono::high_resolution_clock::now();

    chrono::duration<float, std::milli> duration_ms = end_cpu - start_cpu;

    printf("multMatrixOnGPU2d1d <<<(%d,%d), (%d,%d)>>> elapsed %f ms\n", grid.x,
           grid.y,
           block.x, block.y, duration_ms.count());

    // SAFE_CALL kernel error
    SAFE_CALL(hipGetLastError(), "Error with last error");

    // copy kernel result back to host side
    SAFE_CALL(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost), "Error copying d_MatC");
    printMatrix(gpuRef, nx, ny);
    // free device global memory
    SAFE_CALL(hipFree(d_MatA), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatB), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatC), "Error freeing memory");

    // free host memory
    free(h_A);
    free(h_B);
    free(gpuRef);

    // reset device
    SAFE_CALL(hipDeviceReset(), "Error reseting");

    return (0);
}
