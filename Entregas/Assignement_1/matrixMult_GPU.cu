#include "custom.h"
#include "multMatrixOnGPU2d1d.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <math.h>
//#include <hip/hip_fp16.h>
#include <chrono>

#define N0  100
#define N1  200
#define N2  400

using namespace std;

int main(int argc, char **argv){

    int test_n[3];
    test_n[0] = N0;
    test_n[1] = N1;
    test_n[2] = N2;

    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev), "Error device prop");
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    SAFE_CALL(hipSetDevice(dev), "Error setting device");


    for (int i = 0; i < 3; i++) {
      // set up data size of matrix
      int nx = test_n[i];
      int ny = test_n[i];

      int nxy = nx * ny;
      int nBytes = nxy * sizeof(float);
      printf("Matrix size: nx %d ny %d\n", nx, ny);

      // malloc host memory
      float *h_A, *h_B, *gpuRef;
      h_A = (float *)malloc(nBytes);
      h_B = (float *)malloc(nBytes);
      gpu_R = (float *)malloc(nBytes);

      // initialize data at host side

      initialData(h_A, nxy);
      initialData(h_B, nxy);

      // malloc device global memory
      float *d_MatA, *d_MatB, *d_MatC;
      SAFE_CALL(hipMalloc((void **)&d_MatA, nBytes), "Error allocating d_MatA");
      SAFE_CALL(hipMalloc((void **)&d_MatB, nBytes), "Error allocating d_MatB");
      SAFE_CALL(hipMalloc((void **)&d_MatC, nBytes), "Error allocating d_MatC");

      // transfer data from host to device
      SAFE_CALL(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice), "Error copying d_MatA");
      SAFE_CALL(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice), "Error copying d_MatB");

      // invoke kernel at host side
      int dimx = 128 * ((nx + 128 -1) / 128);
      dim3 block(dimx, 1);
      dim3 grid((nx + block.x - 1) / block.x, ny);

      int iterations = 100;

      /**********************************************MULT ON GPU START*****************************************************************************/
      float avTime_gpu = 0.0;
      for (int i = 0; i < iterations; i++) {
        SAFE_CALL(hipMemset(d_MatC, 0, nBytes), "Error setting d_MatC to 0");
        auto start_cpu =  chrono::high_resolution_clock::now();
        multMatrixOnGPU2d1d<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
        SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel");
        auto end_cpu =  chrono::high_resolution_clock::now();
        chrono::duration<float, std::milli> duration_ms = end_cpu - start_cpu;

        avTime_gpu += duration_ms.count();
      }

      avTime_gpu = avTime_gpu / iterations;
      printf("Average time for %d multiplications in GPU with a matrix of %d x %d is %f ms\n", iterations, nx, ny, avTime_gpu);
      /**********************************************MULT ON GPU END*******************************************************************************/

      // SAFE_CALL kernel error
      SAFE_CALL(hipGetLastError(), "Error with last error");

      // copy kernel result back to host side
      SAFE_CALL(hipMemcpy(gpu_R, d_MatC, nBytes, hipMemcpyDeviceToHost), "Error copying d_MatC");

      // free device global memory
      SAFE_CALL(hipFree(d_MatA), "Error freeing memory");
      SAFE_CALL(hipFree(d_MatB), "Error freeing memory");
      SAFE_CALL(hipFree(d_MatC), "Error freeing memory");

      // free host memory
      free(h_A);
      free(h_B);
      free(gpu_R);

      printf("\n\n" );
    }

    // reset device
    SAFE_CALL(hipDeviceReset(), "Error reseting");

    return (0);
}
