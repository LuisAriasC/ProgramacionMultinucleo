#include "hip/hip_runtime.h"
/*
  Author: Luis Carlos Arias Camacho
  Student ID: A01364808
*/
#include <iostream>
#include <cstdio>
#include <cmath>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "common.h"
#include <hip/hip_runtime.h>

#define img_dest "Images/"
#define default_image "dog1.jpeg"
#define C_SIZE 256

using namespace std;

// input - input image one dimensional array
// ouput - output image one dimensional array
// width, height - width and height of the images
// colorWidthStep - number of color bytes (cols * colors)
// grayWidthStep - number of gray bytes
__global__ void bgr_to_gray_kernel(unsigned char* input, unsigned char* output, int width, int height, int colorWidthStep, int grayWidthStep){
	// 2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	if ((xIndex < width) && (yIndex < height)){
		const int color_tid = yIndex * colorWidthStep + (3 * xIndex);
		const int gray_tid = yIndex * grayWidthStep + xIndex;
		const unsigned char blue = input[color_tid];
		const unsigned char green = input[color_tid + 1];
		const unsigned char red = input[color_tid + 2];
		const float gray = red * 0.3f + green * 0.59f + blue * 0.11f;
		output[gray_tid] = static_cast<unsigned char>(gray);
	}
}


__global__ void equalize_image_kernel(unsigned char* output, int* histo,int width, int height, int grayWidthStep){

  __shared__ int n_histo[C_SIZE];

	// 2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
  const int x = threadIdx.x;
  const int y = threadIdx.y;
  const int step_x = blockDim.x;

  int sizeImage = width * height;

	if ((xIndex < width) && (yIndex < height)){
    int h_index = (y * step_x) + x;
    int o_index = (yIndex * grayWidthStep) + xIndex;

    n_histo[h_index] = 0;
    __syncthreads();

    const int tid = yIndex * grayWidthStep + xIndex;
    atomicAdd(&histo[(int)output[tid]], 1);
    __syncthreads();

    //Normalized histogram
    int i;
    for (i = 0; i <= h_index; i++)
      n_histo[h_index] += histo[i];
    __syncthreads();

    /*
    unsigned int aux = (n_histo[h_index] * C_SIZE) / sizeImage;
    n_histo[h_index] = aux;
    __syncthreads();

    output[o_index] = n_histo[h_index];
    */
    if (o_index == 0) {
      for (int i = 0; i < C_SIZE; i++) {
        cout << n_histo[i]  << " "; 
      }
    }
	}
}

void convert_to_gray(const cv::Mat& input, cv::Mat& output, string imageName){


	size_t colorBytes = input.step * input.rows;
	size_t grayBytes = output.step * output.rows;
  //int imSize = input.cols * input.rows;

	unsigned char *d_input, *d_output;
  int * d_histogram;
  int * histogram = (int *)malloc(C_SIZE * sizeof(int));

  for (int i = 0; i < C_SIZE; i++)
    histogram[i] = 0;

	// Allocate device memory
	SAFE_CALL(hipMalloc<unsigned char>(&d_input, colorBytes), "CUDA Malloc Failed");
	SAFE_CALL(hipMalloc<unsigned char>(&d_output, grayBytes), "CUDA Malloc Failed");
  SAFE_CALL(hipMalloc<int>(&d_histogram, C_SIZE * sizeof(int)), "CUDA Malloc Failed");

	// Copy data from OpenCV input image to device memory
	SAFE_CALL(hipMemcpy(d_input, input.ptr(), colorBytes, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");
  SAFE_CALL(hipMemset(d_histogram, 0, C_SIZE * sizeof(int)), "Error setting d_MatC to 0");

  const dim3 block(16, 16);
	const dim3 grid((int)ceil((float)input.cols / block.x), (int)ceil((float)input.rows/ block.y));

	// Launch the color conversion kernel
	bgr_to_gray_kernel <<<grid, block >>>(d_input, d_output, input.cols, input.rows, static_cast<int>(input.step), static_cast<int>(output.step));
  // Synchronize to check for any kernel launch errors
	SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");
  SAFE_CALL(hipMemcpy(output.ptr(), d_output, grayBytes, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");
  //Write the black & white image
  cv::imwrite("Images/bw_" + imageName , output);

  equalize_image_kernel<<<grid, block >>>(d_output, d_histogram, input.cols, input.rows, static_cast<int>(output.step));
  // Synchronize to check for any kernel launch errors
	SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");
  SAFE_CALL(hipMemcpy(histogram, d_histogram, C_SIZE * sizeof(int), hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");
  //Write the black & white image
  cv::imwrite("Images/eq_gpu_" + imageName , output);

  /*
  int sum = 0;
  for (int i = 0; i < C_SIZE; i++)
    sum += histogram[i];
    //printf("%d : %d\n", i, histogram[i]);
  printf("%d : %d\n", imSize, sum);
*/
	// Free the device memory
	SAFE_CALL(hipFree(d_input), "CUDA Free Failed");
	SAFE_CALL(hipFree(d_output), "CUDA Free Failed");
}

void equalizer_cpu(const cv::Mat &input, cv::Mat &output, string imageName){

  int width = input.cols;
  int height = input.rows;
  int size_ = width * height;

  //Histogram
  int histo[C_SIZE]{};

  //Fill histogram
  for (int i = 0; i < size_; i++)
    histo[input.ptr()[i]]++;

  //Normalized histogram
  long n_histo[C_SIZE]{};
  for (int i = 0; i < C_SIZE; i++){
      for(int j = 0; j <= i; j++)
          n_histo[i] += histo[j];
      unsigned int aux  = (n_histo[i]*C_SIZE) / size_;
      n_histo[i] = aux;
  }

  for (int i = 0; i < size_; i++)
    output.ptr()[i] = n_histo[input.ptr()[i]];

  cv::imwrite("Images/eq_" + imageName , output);
}

int main(int argc, char *argv[]){

	string inputImage;

	if(argc < 2)
		inputImage = default_image;
  else
  	inputImage = argv[1];

	// Read input image from the disk
	cv::Mat input = cv::imread(img_dest + inputImage, CV_LOAD_IMAGE_COLOR);

	if (input.empty()){
		cout << "Image Not Found!" << std::endl;
		cin.get();
		return -1;
	}

	//Create output image
	cv::Mat output(input.rows, input.cols, CV_8UC1);
  //Create equalized output image
  cv::Mat eq_output(input.rows, input.cols, CV_8UC1);

	//Convert image to gray
	convert_to_gray(input, output, inputImage);
  //equalizer_cpu(output, eq_output, inputImage);

	//Allow the windows to resize
  /*
	namedWindow("Input", cv::WINDOW_NORMAL);
	namedWindow("Output", cv::WINDOW_NORMAL);

	//Show the input and output
	imshow("Input", input);
	imshow("Output", output);

	//Wait for key press
	cv::waitKey();
  */
	return 0;
}
