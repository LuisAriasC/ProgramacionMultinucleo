#include "hip/hip_runtime.h"
/*
  Author: Luis Carlos Arias Camacho
  Student ID: A01364808
*/
#include <iostream>
#include <cstdio>
#include <cmath>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "common.h"
#include <hip/hip_runtime.h>

#define img_dest "Images/"
#define default_image "dog1.jpeg"
#define C_SIZE 256

using namespace std;

int * equalize(int * histogram, int size){
    int step = size / C_SIZE;
    int sum = 0;
    int * n_histogram = (int * )calloc(C_SIZE,sizeof(int));

    for(int i=0; i < C_SIZE; i++){
        sum += histogram[i];
        n_histogram[i] = sum / step;
    }
    return n_histogram;
}

// input - input image one dimensional array
// ouput - output image one dimensional array
// width, height - width and height of the images
// colorWidthStep - number of color bytes (cols * colors)
// grayWidthStep - number of gray bytes
__global__ void bgr_to_gray_kernel(unsigned char* input, unsigned char* output, int width, int height, int colorWidthStep, int grayWidthStep){
	// 2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	if ((xIndex < width) && (yIndex < height)){
		const int color_tid = yIndex * colorWidthStep + (3 * xIndex);
		const int gray_tid = yIndex * grayWidthStep + xIndex;
		const unsigned char blue = input[color_tid];
		const unsigned char green = input[color_tid + 1];
		const unsigned char red = input[color_tid + 2];
		const float gray = red * 0.3f + green * 0.59f + blue * 0.11f;
		output[gray_tid] = static_cast<unsigned char>(gray);
	}
}

void equalizer_cpu(const cv::Mat &input, cv::Mat &output, string imageName){

  int width = input.cols;
  int height = input.rows;
  int size_ = width * height;

  //Histogram
  int histo[C_SIZE]{};

  //Fill histogram
  for (int i = 0; i < size_; i++)
    histo[input.ptr()[i]]++;

  //Normalize
  int step = size_ / C_SIZE;
  int sum = 0;
  int n_histo[C_SIZE]{};
  for(int i=0; i < C_SIZE; i++){
      sum += histo[i];
      n_histo[i] = sum / step;
  }

  for (int i = 0; i < size_; i++)
    output.ptr()[i] = n_histo[input.ptr()[i]];

  cv::imwrite("Images/eq_cpu_" + imageName , output);
}



__global__ void get_histogram_kernel(unsigned char* output, int* histo,int width, int height, int grayWidthStep){

	// 2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	if ((xIndex < width) && (yIndex < height)){
    const int tid = yIndex * grayWidthStep + xIndex;
    atomicAdd(&histo[(int)output[tid]], 1);
    __syncthreads();
	}
}

__global__ void set_image_kernel(unsigned char* input,unsigned char* output, int * histogram, int width, int height, int step){

    __shared__ int * shHistogram;
    for(int i = 0;i<256;i++){
        shHistogram[i] = histogram[i];
    }
    __syncthreads();

    const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if ((xIndex < width) && (yIndex < height)){
        const int tid = yIndex * step + xIndex;
        output[tid] =static_cast<unsigned char>(shHistogram[input[tid]]);
    }
}

void convert_to_gray(const cv::Mat& input, cv::Mat& output, cv::Mat& eq_output, string imageName){


	size_t colorBytes = input.step * input.rows;
	size_t grayBytes = output.step * output.rows;
  int imSize = input.cols * input.rows;

	unsigned char *d_input, *d_output, *de_output;
  int * d_histogram;
  int * histogram = (int *)malloc(C_SIZE * sizeof(int));
  for (int i = 0; i < C_SIZE; i++)
    histogram[i] = 0;

	// Allocate device memory
	SAFE_CALL(hipMalloc<unsigned char>(&d_input, colorBytes), "CUDA Malloc Failed");
	SAFE_CALL(hipMalloc<unsigned char>(&d_output, grayBytes), "CUDA Malloc Failed");
  SAFE_CALL(hipMalloc<unsigned char>(&de_output, grayBytes), "CUDA Malloc Failed");
  SAFE_CALL(hipMalloc<int>(&d_histogram, C_SIZE * sizeof(int)), "CUDA Malloc Failed");

	// Copy data from OpenCV input image to device memory
	SAFE_CALL(hipMemcpy(d_input, input.ptr(), colorBytes, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");
  SAFE_CALL(hipMemset(d_histogram, 0, C_SIZE * sizeof(int)), "Error setting d_MatC to 0");

  const dim3 block(16, 16);
	const dim3 grid((int)ceil((float)input.cols / block.x), (int)ceil((float)input.rows/ block.y));

	// Launch the color conversion kernel
	bgr_to_gray_kernel <<<grid, block >>>(d_input, d_output, input.cols, input.rows, static_cast<int>(input.step), static_cast<int>(output.step));
  // Synchronize to check for any kernel launch errors
	SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");
  SAFE_CALL(hipMemcpy(output.ptr(), d_output, grayBytes, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");
  //Write the black & white image
  cv::imwrite("Images/bw_" + imageName , output);

  printf("In CPU\n");
  equalizer_cpu(output, eq_output, imageName);
  printf("END CPU\n");

  get_histogram_kernel<<<grid, block >>>(d_output, d_histogram, input.cols, input.rows, static_cast<int>(output.step));
  // Synchronize to check for any kernel launch errors
	SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");
  SAFE_CALL(hipMemcpy(histogram, d_histogram, C_SIZE * sizeof(int), hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");

  int * f_histogram = equalize(histogram, imSize);

  int sum = 0;
  for (int i = 0; i < C_SIZE; i++)
    sum += histogram[i];
  printf("%d : %d\n", imSize, sum);

  for (int i = 0; i < C_SIZE; i++)
    printf("%d : %d\n", i, f_histogram[i]);

  set_image_kernel<<<grid, block>>>(d_output, de_output, f_histogram, output.cols, output.rows, static_cast<int>(output.step));
  // Synchronize to check for any kernel launch errors
  SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");
  SAFE_CALL(hipMemcpy(eq_output.ptr(), de_output, grayBytes, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");
  //Write the black & white image
  cv::imwrite("Images/eq_gpu_" + imageName , eq_output);

  //Write the black & white image
  //cv::imwrite("Images/eq_gpu_" + imageName , output);

	// Free the device memory
	SAFE_CALL(hipFree(d_input), "CUDA Free Failed");
	SAFE_CALL(hipFree(d_output), "CUDA Free Failed");
}

int main(int argc, char *argv[]){

	string inputImage;

	if(argc < 2)
		inputImage = default_image;
  else
  	inputImage = argv[1];

	// Read input image from the disk
	cv::Mat input = cv::imread(img_dest + inputImage, CV_LOAD_IMAGE_COLOR);

	if (input.empty()){
		cout << "Image Not Found!" << std::endl;
		cin.get();
		return -1;
	}

	//Create output image
	cv::Mat output(input.rows, input.cols, CV_8UC1);
  //Create equalized output image
  cv::Mat eq_output(input.rows, input.cols, CV_8UC1);

	//Convert image to gray
	convert_to_gray(input, output, eq_output, inputImage);
  //equalizer_cpu(output, eq_output, inputImage);

	//Allow the windows to resize
  /*
	namedWindow("Input", cv::WINDOW_NORMAL);
	namedWindow("Output", cv::WINDOW_NORMAL);

	//Show the input and output
	imshow("Input", input);
	imshow("Output", output);

	//Wait for key press
	cv::waitKey();
  */
	return 0;
}
